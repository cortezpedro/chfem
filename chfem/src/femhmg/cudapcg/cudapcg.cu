#include "hip/hip_runtime.h"
/*
  =====================================================================
  Universidade Federal Fluminense (UFF) - Niteroi, Brazil
  Institute of Computing
  Authors: Cortez Lopes, P., Pereira., A.
  contact: pedrocortez@id.uff.br
  =====================================================================
*/

#include "cudapcg.h"
#include "error_handling.h"
#include "kernels/wrappers.h"

#include "solvers/cg.h"
#include "solvers/pcg.h"
#include "solvers/cg3.h"
#include "solvers/pcg3.h"

#include "solvers/minres.h"
#include "solvers/pminres.h"
#include "solvers/minres3.h"
#include "solvers/pminres3.h"

// Only available for permeability analysis for now.
#include "solvers/cg2.h"
#include "solvers/pcg2.h"
#include "solvers/minres2.h"
#include "solvers/pminres2.h"

//---------------------------------
///////////////////////////////////
//////////// GLOBALS //////////////
///// (FOR INTERNAL USE ONLY) /////
///////////////////////////////////
//---------------------------------

cudapcgSolver_t *solver = NULL;

//---------------------------------
///////////////////////////////////
//////// PRIVATE FUNCTIONS ////////
////////// (DECLARATIONS) /////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setAnalysis();
cudapcgFlag_t setSolver(unsigned int flag);
cudapcgFlag_t setResNorm(cudapcgFlag_t flag);
//------------------------------------------------------------------------------

//---------------------------------
///////////////////////////////////
//////// PRIVATE FUNCTIONS ////////
//////////  (AUXILIARY)  //////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setModelStruct_simple(cudapcgModel_t **ptr, const void *data){
  if (data==NULL) return CUDAPCG_FALSE;
  *ptr = (cudapcgModel_t *)data;
  return CUDAPCG_TRUE;
}
cudapcgFlag_t (*setModelStruct)(cudapcgModel_t **, const void *) = setModelStruct_simple;
//------------------------------------------------------------------------------
void freeModelStruct(cudapcgModel_t *model){
    if (solver->model->name !=NULL)            {free(solver->model->name); solver->model->name = NULL;}
    if (solver->model->image!=NULL)            {HANDLE_ERROR(hipFree(solver->model->image));            solver->model->image=NULL;}
    if (solver->model->parametric_density_field!=NULL){HANDLE_ERROR(hipFree(solver->model->parametric_density_field)); solver->model->parametric_density_field=NULL;}
    if (solver->model->pore_map!=NULL)         {HANDLE_ERROR(hipFree(solver->model->pore_map));         solver->model->pore_map=NULL;}
    if (solver->model->border_pore_map!=NULL)  {HANDLE_ERROR(hipFree(solver->model->border_pore_map));  solver->model->border_pore_map=NULL;}    
    if (solver->model->periodic2DOF_map!=NULL) {HANDLE_ERROR(hipFree(solver->model->periodic2DOF_map)); solver->model->periodic2DOF_map=NULL;}
    if (solver->model->DOF2periodic_map!=NULL) {HANDLE_ERROR(hipFree(solver->model->DOF2periodic_map)); solver->model->DOF2periodic_map=NULL;}
    free(solver->model); solver->model=NULL;
    return;
}
//------------------------------------------------------------------------------
cudapcgFlag_t isModelValid(cudapcgSolver_t *_solver){
  if (_solver==NULL) return CUDAPCG_FALSE;
  if (_solver->model==NULL) return CUDAPCG_FALSE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------

//---------------------------------
///////////////////////////////////
//////// PUBLIC FUNCTIONS /////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
// Setters and getters
cudapcgFlag_t cudapcgSetNumTol(cudapcgTol_t t){ if(solver==NULL) return CUDAPCG_FALSE; solver->num_tol = t; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetMaxIterations(unsigned int n){ if(solver==NULL) return CUDAPCG_FALSE; solver->max_iterations = n; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetXReduceMode(cudapcgFlag_t flag){ if(solver==NULL) return CUDAPCG_FALSE; solver->xreduce_flag = flag <= CUDAPCG_XREDUCE_FULL ? flag : CUDAPCG_XREDUCE_FULL; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetXReduceShift(unsigned int shift){ if(solver==NULL) return CUDAPCG_FALSE; solver->xreduce_shift = shift; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetXReduceScale(double scl){ if(solver==NULL) return CUDAPCG_FALSE; solver->xreduce_scale = scl; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetReduceStabFactor(double scl){ if(solver==NULL) return CUDAPCG_FALSE; solver->reduce_stab_factor = scl; return CUDAPCG_TRUE; }
cudapcgFlag_t cudapcgSetSolver(cudapcgFlag_t flag){ if(solver==NULL) return CUDAPCG_FALSE; return setSolver(flag);}
cudapcgFlag_t cudapcgSetPreconditioner(cudapcgFlag_t flag){ if(solver==NULL) return CUDAPCG_FALSE; solver->preconditioner_flag = (flag>0) ? CUDAPCG_TRUE : CUDAPCG_FALSE; solver->solver_flag++; return setSolver(solver->solver_flag-1);}
cudapcgFlag_t cudapcgSetResNorm(cudapcgFlag_t flag){ if(solver==NULL) return CUDAPCG_FALSE; return setResNorm(flag);}
unsigned int cudapcgGetNumIterations(){ if(solver==NULL) return 0; return solver->iteration; }
unsigned int cudapcgGetMaxNumIterations(){ if(solver==NULL) return 0; return solver->max_iterations; }
cudapcgVar_t cudapcgGetResidual(){ if(solver==NULL) return 0; return solver->residual; }
//---------------------------------------------------------------------------------

cudapcgFlag_t cudapcgInit(cudapcgFlag_t analysis_flag, cudapcgFlag_t parStrategy_flag){

    // Make sure that no previously allocated data messes with this init -- for safety!
    cudapcgEnd();

    // Initial checks for potential inconsistencies in provided model
    if (analysis_flag != CUDAPCG_THERMAL_2D           && analysis_flag != CUDAPCG_THERMAL_3D &&
        analysis_flag != CUDAPCG_ELASTIC_2D           && analysis_flag != CUDAPCG_ELASTIC_3D &&
        analysis_flag != CUDAPCG_FLUID_2D             && analysis_flag != CUDAPCG_FLUID_3D)
        return CUDAPCG_FALSE;

    // Check if user provided flag for unsupported functionalities
    if ((analysis_flag == CUDAPCG_FLUID_2D || analysis_flag == CUDAPCG_FLUID_3D) && parStrategy_flag != CUDAPCG_NBN){
      if (parStrategy_flag == CUDAPCG_EBE){ // ElemByElem
        printf("ERROR: EBE solver is unavailable for FLUID analysis.\n");
      } else {
        printf("ERROR: Invalid \"parStrategy_flag\" (%d) for FLUID analysis.\n",parStrategy_flag);
      }
      return CUDAPCG_FALSE;
    }

    solver = (cudapcgSolver_t *)malloc(sizeof(cudapcgSolver_t));
    if (solver == NULL){
        printf("ERROR: Memory allocation for solver struct has failed.\n");
        return CUDAPCG_FALSE;
    }

    solver->count = 0;

    solver->header_str = (char *)malloc(sizeof(char));
    strcpy(solver->header_str,"");

    solver->model = NULL;

    solver->analysis_flag = analysis_flag;
    solver->parallelStrategy_flag = parStrategy_flag;

    solver->residual = 1.0;
    solver->iteration = 0;
    solver->total_time = 0.0;
    solver->mean_time_per_iteration = 0.0;
    solver->foundSolution_flag = CUDAPCG_FALSE;

    solver->num_tol = CUDAPCG_TOLERANCE;
    solver->max_iterations = CUDAPCG_MAX_ITERATIONS;

    solver->xreduce_flag = CUDAPCG_XREDUCE_FULL;
    solver->xreduce_shift = 0;
    solver->xreduce_scale = 1.0;
    solver->reduce_stab_factor = 0.0001;

    solver->resnorm_flag = CUDAPCG_L2_NORM;

    solver->userAllocatedArrays_flag = CUDAPCG_FALSE;
    solver->x0_hasBeenSet_flag = CUDAPCG_FALSE;
    solver->mustAssemblePreConditioner = CUDAPCG_FALSE;

    if (!setAnalysis())
      return CUDAPCG_FALSE;

    // Initialize solver
    solver->solver_flag = CUDAPCG_CG_SOLVER;
    solver->preconditioner_flag = CUDAPCG_TRUE;
    solver->solve = solve_pcg;
    solver->setX0 = setX0_cg;
    solver->allocDeviceArrays = allocDeviceArrays_cg;
    solver->freeDeviceArrays = freeDeviceArrays_cg;

    // Initialize arrays pointing to NULL
    solver->x = NULL;
    solver->r = NULL;
    solver->d = NULL;
    solver->q = NULL;
    solver->s = NULL; // used in minres
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgEnd(){
    if (solver == NULL) return CUDAPCG_FALSE;
    free(solver->header_str);
    if (solver->model!=NULL && solver->model->freeAllowed_flag) freeModelStruct(solver->model);
    if (solver->userAllocatedArrays_flag) cudapcgFreeArrays();
    if (solver->x!=NULL) HANDLE_ERROR(hipFree(solver->x)); solver->x=NULL;
    if (solver->r!=NULL) HANDLE_ERROR(hipFree(solver->r)); solver->r=NULL;
    if (solver->d!=NULL) HANDLE_ERROR(hipFree(solver->d)); solver->d=NULL;
    if (solver->q!=NULL) HANDLE_ERROR(hipFree(solver->q)); solver->q=NULL;
    if (solver->s!=NULL) HANDLE_ERROR(hipFree(solver->s)); solver->s=NULL;
    #if defined CUDAPCG_MATKEY_32BIT || defined CUDAPCG_MATKEY_64BIT
      freeLocalK();
    #endif
    free(solver);
    solver = NULL;
    // Reset device (call from cuda API) -- for safety!
    hipDeviceReset();
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgModel_t * cudapcgNewModel(void){
  cudapcgModel_t *new_model = (cudapcgModel_t *)malloc(sizeof(cudapcgModel_t));
  if (!new_model) return NULL;
  new_model->name = NULL;
  new_model->freeAllowed_flag = CUDAPCG_TRUE;
  new_model->parStrategy_flag = CUDAPCG_NBN;
  new_model->poremap_flag     = CUDAPCG_POREMAP_NUM;
  new_model->parametric_density_field_flag = CUDAPCG_FALSE;
  #ifdef TESTING_STENCIL
  new_model->SBS_flag = CUDAPCG_TRUE;
  #else
  new_model->SBS_flag = CUDAPCG_FALSE;
  #endif
  new_model->nrows = 0;
  new_model->ncols = 0;
  new_model->nlayers = 0;
  new_model->nelem = 0;
  new_model->nvars = 0;
  new_model->nkeys = 0;
  new_model->localmtxdim = 0;
  new_model->nporenodes = 0;
  new_model->nbordernodes = 0;
  new_model->image = NULL;
  new_model->parametric_density_field = NULL;
  new_model->limits_density_field[0]=0.0;
  new_model->limits_density_field[1]=0.0;
  new_model->pore_map = NULL;
  new_model->border_pore_map = NULL;
  new_model->periodic2DOF_map = NULL;
  new_model->DOF2periodic_map = NULL;
  return new_model;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetModel(cudapcgModel_t *model){
  if (solver==NULL) return CUDAPCG_FALSE;
  if (solver->model!=NULL && solver->model->freeAllowed_flag)
    freeModelStruct(solver->model);
  solver->model = model;
  cudapcgFreeArrays();
  size_t var_sz = sizeof(cudapcgVar_t)*((size_t)solver->model->nvars);
  if (solver->r!=NULL) HANDLE_ERROR(hipFree(solver->r)); solver->r=NULL;
  HANDLE_ERROR(hipMalloc(&(solver->r),var_sz));
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetModelConstructorFcn(cudapcgFlag_t (*fcn)(cudapcgModel_t **, const void *)){
  if (fcn==NULL) return CUDAPCG_FALSE;
  setModelStruct = fcn;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgBuildModel(const void *data){
  if (solver==NULL) return CUDAPCG_FALSE;
  if (solver->model==NULL){
    solver->model = cudapcgNewModel();
  } else if (solver->model->freeAllowed_flag){
    freeModelStruct(solver->model);
    solver->model = cudapcgNewModel();
  }
  if (!setModelStruct(&(solver->model),data)) return CUDAPCG_FALSE;
  cudapcgFreeArrays();
  size_t var_sz = sizeof(cudapcgVar_t)*((size_t)solver->model->nvars);
  if (solver->r!=NULL) HANDLE_ERROR(hipFree(solver->r)); solver->r=NULL;
  HANDLE_ERROR(hipMalloc(&(solver->r),var_sz));
  if (solver->analysis_flag < CUDAPCG_FLUID_2D){
    if (solver->model->parametric_density_field_flag == CUDAPCG_FALSE){
      if (solver->model->image==NULL)
        HANDLE_ERROR(hipMalloc(&(solver->model->image),sizeof(cudapcgMap_t)*solver->model->nelem));
    } else {
      if (solver->analysis_flag >= CUDAPCG_ELASTIC_2D){ // not required for thermal analysis
        HANDLE_ERROR(hipMalloc(&(solver->model->image),sizeof(cudapcgMap_t)*solver->model->nelem));
      }
      HANDLE_ERROR(hipMalloc(&(solver->model->parametric_density_field),sizeof(parametricScalarField_t)*solver->model->nelem));
    }
  } else {
    if (solver->model->periodic2DOF_map==NULL)
      HANDLE_ERROR(hipMalloc(&(solver->model->periodic2DOF_map),sizeof(cudapcgIdMap_t)*solver->model->nelem));
    if (solver->model->poremap_flag == CUDAPCG_POREMAP_IMG){
        if (solver->model->pore_map==NULL)
          HANDLE_ERROR(hipMalloc(&(solver->model->pore_map),sizeof(cudapcgFlag_t)*solver->model->nelem));
    } else if (solver->model->poremap_flag == CUDAPCG_POREMAP_NUM){
        if (solver->model->border_pore_map==NULL)
          HANDLE_ERROR(hipMalloc(&(solver->model->border_pore_map),sizeof(cudapcgFlag_t)*solver->model->nbordernodes));
        if (solver->model->DOF2periodic_map==NULL)
          HANDLE_ERROR(hipMalloc(&(solver->model->DOF2periodic_map),sizeof(cudapcgIdMap_t)*(solver->model->nporenodes+solver->model->nbordernodes)));
    }
  }
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetX0(cudapcgVar_t *x0, cudapcgFlag_t mustInterpolate){
    if (!isModelValid(solver)) return CUDAPCG_FALSE;
    // Check if an array was provided
    if (x0 != NULL) return solver->setX0(solver,x0,mustInterpolate);
    // Recieved null pointer for x0
    if (!solver->x0_hasBeenSet_flag) return CUDAPCG_TRUE;
    solver->x0_hasBeenSet_flag = CUDAPCG_FALSE;
    cudapcgVar_t **ptr_to_device_x0 = solver->solver_flag < CUDAPCG_CG2_SOLVER ? &(solver->x) : &(solver->d);
    if (*ptr_to_device_x0 == NULL) return CUDAPCG_TRUE;
    if (solver->userAllocatedArrays_flag){
        zeros(*ptr_to_device_x0,solver->model->nvars);
        return CUDAPCG_TRUE;
    }
    if (*ptr_to_device_x0!=NULL) HANDLE_ERROR(hipFree(*ptr_to_device_x0)); *ptr_to_device_x0 = NULL;
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetRHS(cudapcgVar_t * RHS){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    HANDLE_ERROR(hipMemcpy(solver->r,RHS,solver->model->nvars*sizeof(cudapcgVar_t),hipMemcpyHostToDevice));
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetImage(cudapcgMap_t *img){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    // Obs.: Arr size always works with solver->model->nelem because it is numerically equivalent to (valid_nodes/dof_per_node)
    HANDLE_ERROR(hipMemcpy(solver->model->image,img,solver->model->nelem*sizeof(cudapcgMap_t),hipMemcpyHostToDevice));
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetParametricDensityField(parametricScalarField_t *field, double fmin, double fmax){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    HANDLE_ERROR(hipMemcpy(solver->model->parametric_density_field,field,solver->model->nelem*sizeof(parametricScalarField_t),hipMemcpyHostToDevice));
    solver->model->limits_density_field[0] = fmin;
    solver->model->limits_density_field[1] = fmax;
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetPoreMap(cudapcgFlag_t *pores){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    if (solver->model->poremap_flag == CUDAPCG_POREMAP_IMG){
    // Obs.: Arr size always works with solver->model->nelem because it is numerically equivalent to (valid_nodes/dof_per_node)
    HANDLE_ERROR(hipMemcpy(solver->model->pore_map,pores,solver->model->nelem*sizeof(cudapcgFlag_t),hipMemcpyHostToDevice));
    } else if (solver->model->poremap_flag == CUDAPCG_POREMAP_NUM){
    HANDLE_ERROR(hipMemcpy(solver->model->border_pore_map,pores,solver->model->nbordernodes*sizeof(cudapcgFlag_t),hipMemcpyHostToDevice));
    } else return CUDAPCG_FALSE;
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetPeriodic2DOFMap(cudapcgIdMap_t *vars){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    // if (solver->model->poremap_flag != CUDAPCG_POREMAP_NUM) return CUDAPCG_FALSE;
    // Obs.: Arr size always works with solver->model->nelem because it is numerically equivalent to (valid_nodes/dof_per_node)
    HANDLE_ERROR(hipMemcpy(solver->model->periodic2DOF_map,vars,solver->model->nelem*sizeof(cudapcgIdMap_t),hipMemcpyHostToDevice));
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetDOF2PeriodicMap(cudapcgIdMap_t *nodes){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    if (solver->model->poremap_flag != CUDAPCG_POREMAP_NUM) return CUDAPCG_FALSE;
    HANDLE_ERROR(hipMemcpy(solver->model->DOF2periodic_map,nodes,(solver->model->nporenodes+solver->model->nbordernodes)*sizeof(cudapcgIdMap_t),hipMemcpyHostToDevice));
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetLclMtxs(cudapcgVar_t * LclMtxs){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    size_t sz = solver->model->nkeys*solver->model->localmtxdim*sizeof(cudapcgVar_t);
    #if defined CUDAPCG_MATKEY_32BIT || defined CUDAPCG_MATKEY_64BIT
      freeLocalK(); // for safety
      allocLocalK(sz);
    #endif
    setLocalK(LclMtxs,sz);
    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgAllocateArrays(){
    if (!isModelValid(solver))
        return CUDAPCG_FALSE;
    return solver->allocDeviceArrays(solver);
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgFreeArrays(){
    if (solver == NULL)
        return CUDAPCG_FALSE;
    return solver->freeDeviceArrays(solver);
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSolve(cudapcgVar_t *res_x){
    if (!isModelValid(solver) || res_x==NULL)
        return CUDAPCG_FALSE;

    double t = omp_get_wtime();

    cudapcgFlag_t mustFreeAfterPCG = CUDAPCG_TRUE;
    if (solver->userAllocatedArrays_flag)
        mustFreeAfterPCG = CUDAPCG_FALSE;
    else
        solver->allocDeviceArrays(solver);

    solver->solve(solver,res_x);

    if (mustFreeAfterPCG)
        solver->freeDeviceArrays(solver);

    solver->total_time = omp_get_wtime()-t;

    return solver->foundSolution_flag;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgSetHeaderString(char *header){
    if (solver==NULL)
      return CUDAPCG_FALSE;

    free(solver->header_str);

    if (header==NULL){
      solver->header_str = (char *)malloc(sizeof(char));
      strcpy(solver->header_str,"");
      return CUDAPCG_TRUE;
    }

    solver->header_str = (char *)malloc(sizeof(char)*(strlen(header)+1));
    if (solver->header_str==NULL){
      solver->header_str = (char *)malloc(sizeof(char));
      strcpy(solver->header_str,"");
      return CUDAPCG_FALSE;
    }
    strcpy(solver->header_str,header);

    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgPrintSolverReport(){
    if (solver==NULL)
      return CUDAPCG_FALSE;

    if (solver->header_str) printf("%s",solver->header_str);
    if (solver->foundSolution_flag)
        printf("PCG solver found a solution in %d iterations.\n",solver->iteration);
    else
        printf("PCG solver did not converge within provided max number of iterations (%d).\n",solver->max_iterations);

    if (solver->header_str) printf("%s",solver->header_str);
    printf("Residual: %e\n",solver->residual);

    return cudapcgPrintSolverMetrics();
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgPrintSolverReport2(char *dest){
    if (dest==NULL)
      return CUDAPCG_FALSE;

    int sz = sprintf(
      dest,
      "%sPCG solver ran through %d iterations.\n"\
      "%sResidual: %e\n"\
      "%sMean time per iteration: %.2e s\n"\
      "%sElapsed time: %.2e s\n",
      solver->header_str,solver->iteration,
      solver->header_str,solver->residual,
      solver->header_str,solver->mean_time_per_iteration,
      solver->header_str,solver->total_time
    );

    if (sz<0) return CUDAPCG_FALSE;

    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgPrintSolverMetrics(){
    if (solver==NULL)
      return CUDAPCG_FALSE;

    if (solver->header_str) printf("%s",solver->header_str);
    printf("Mean time per iteration: %.2e s\n",solver->mean_time_per_iteration);

    if (solver->header_str) printf("%s",solver->header_str);
    printf("Elapsed time: %.2e s\n",solver->total_time);

    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t cudapcgPrintSolverMetrics2(char *dest){
    if (dest==NULL)
      return CUDAPCG_FALSE;

    int sz = sprintf(
      dest,
      "%sMean time per iteration: %.2e s\n"\
      "%sElapsed time: %.2e s\n",
      solver->header_str,solver->mean_time_per_iteration,
      solver->header_str,solver->total_time
    );

    if (sz<0) return CUDAPCG_FALSE;

    return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------

//---------------------------------
///////////////////////////////////
//////// PRIVATE FUNCTIONS ////////
//////// (IMPLEMENTATIONS) ////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setAnalysis(){

  if (solver == NULL) return CUDAPCG_FALSE;

  if (solver->analysis_flag == CUDAPCG_THERMAL_2D){

    solver->assemblePreConditioner = assemblePreConditioner_thermal_2D;
    solver->applyPreConditioner = applyPreConditioner_thermal_2D;
    solver->applyinvPreConditioner = applyinvPreConditioner_thermal_2D;
    solver->Aprod = Aprod_thermal_2D;
    solver->PreConditionerAprod = PreConditionerAprod_thermal_2D;
    solver->dotPreConditioner = dotPreConditioner_thermal_2D;
    solver->dotinvPreConditioner = dotinvPreConditioner_thermal_2D;
    solver->dotAprod = dotAprod_thermal_2D;
    solver->dotA2prod = dotA2prod_thermal_2D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_thermal_2D;

  } else if (solver->analysis_flag == CUDAPCG_THERMAL_3D){

    solver->assemblePreConditioner = assemblePreConditioner_thermal_3D;
    solver->applyPreConditioner = applyPreConditioner_thermal_3D;
    solver->applyinvPreConditioner = applyinvPreConditioner_thermal_3D;
    solver->Aprod = Aprod_thermal_3D;
    solver->PreConditionerAprod = PreConditionerAprod_thermal_3D;
    solver->dotPreConditioner = dotPreConditioner_thermal_3D;
    solver->dotinvPreConditioner = dotinvPreConditioner_thermal_3D;
    solver->dotAprod = dotAprod_thermal_3D;
    solver->dotA2prod = dotA2prod_thermal_3D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_thermal_3D;

  } else if (solver->analysis_flag == CUDAPCG_ELASTIC_2D){

    solver->assemblePreConditioner = assemblePreConditioner_elastic_2D;
    solver->applyPreConditioner = applyPreConditioner_elastic_2D;
    solver->applyinvPreConditioner = applyinvPreConditioner_elastic_2D;
    solver->Aprod = Aprod_elastic_2D;
    solver->PreConditionerAprod = PreConditionerAprod_elastic_2D;
    solver->dotPreConditioner = dotPreConditioner_elastic_2D;
    solver->dotinvPreConditioner = dotinvPreConditioner_elastic_2D;
    solver->dotAprod = dotAprod_elastic_2D;
    solver->dotA2prod = dotA2prod_elastic_2D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_elastic_2D;

  } else if (solver->analysis_flag == CUDAPCG_ELASTIC_3D){

    solver->assemblePreConditioner = assemblePreConditioner_elastic_3D;
    solver->applyPreConditioner = applyPreConditioner_elastic_3D;
    solver->applyinvPreConditioner = applyinvPreConditioner_elastic_3D;
    solver->Aprod = Aprod_elastic_3D;
    solver->PreConditionerAprod = PreConditionerAprod_elastic_3D;
    solver->dotPreConditioner = dotPreConditioner_elastic_3D;
    solver->dotinvPreConditioner = dotinvPreConditioner_elastic_3D;
    solver->dotAprod = dotAprod_elastic_3D;
    solver->dotA2prod = dotA2prod_elastic_3D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_elastic_3D;

  } else if (solver->analysis_flag == CUDAPCG_FLUID_2D){

    solver->assemblePreConditioner = NULL;
    solver->applyPreConditioner = applyPreConditioner_fluid_2D;
    solver->applyinvPreConditioner = applyinvPreConditioner_fluid_2D;
    solver->Aprod = Aprod_fluid_2D;
    solver->PreConditionerAprod = PreConditionerAprod_fluid_2D;
    solver->dotPreConditioner = dotPreConditioner_fluid_2D;
    solver->dotinvPreConditioner = dotinvPreConditioner_fluid_2D;
    solver->dotAprod = dotAprod_fluid_2D;
    solver->dotA2prod = dotA2prod_fluid_2D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_fluid_2D;

  } else if (solver->analysis_flag == CUDAPCG_FLUID_3D){

    solver->assemblePreConditioner = NULL;
    solver->applyPreConditioner = applyPreConditioner_fluid_3D;
    solver->applyinvPreConditioner = applyinvPreConditioner_fluid_3D;
    solver->Aprod = Aprod_fluid_3D;
    solver->PreConditionerAprod = PreConditionerAprod_fluid_3D;
    solver->dotPreConditioner = dotPreConditioner_fluid_3D;
    solver->dotinvPreConditioner = dotinvPreConditioner_fluid_3D;
    solver->dotAprod = dotAprod_fluid_3D;
    solver->dotA2prod = dotA2prod_fluid_3D;
    solver->dotPreConditionerA2prod = dotPreConditionerA2prod_fluid_3D;

  } else return CUDAPCG_FALSE;

  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t setSolver(unsigned int flag){

  if (solver == NULL) return CUDAPCG_FALSE;

  if (flag > CUDAPCG_MINRES2_SOLVER){
    printf("ERROR: Invalid \"solverFlag\" (%d).\n",flag);
    return CUDAPCG_FALSE;
  }

  if (solver->solver_flag == flag) return CUDAPCG_TRUE;

  if (flag == CUDAPCG_CG_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pcg;
    else
      solver->solve = solve_cg;
    solver->setX0 = setX0_cg;
    solver->allocDeviceArrays = allocDeviceArrays_cg;
    solver->freeDeviceArrays = freeDeviceArrays_cg;

  } else if (flag == CUDAPCG_MINRES_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pminres;
    else
      solver->solve = solve_minres;
    solver->setX0 = setX0_minres;
    solver->allocDeviceArrays = allocDeviceArrays_minres;
    solver->freeDeviceArrays = freeDeviceArrays_minres;

  } else if (flag == CUDAPCG_CG3_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pcg3;
    else
      solver->solve = solve_cg3;
    solver->setX0 = setX0_cg3;
    solver->allocDeviceArrays = allocDeviceArrays_cg3;
    solver->freeDeviceArrays = freeDeviceArrays_cg3;

  } else if (flag == CUDAPCG_MINRES3_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pminres3;
    else
      solver->solve = solve_minres3;
    solver->setX0 = setX0_minres3;
    solver->allocDeviceArrays = allocDeviceArrays_minres3;
    solver->freeDeviceArrays = freeDeviceArrays_minres3;

  } else if (flag == CUDAPCG_CG2_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pcg2;
    else
      solver->solve = solve_cg2;
    solver->setX0 = setX0_cg2;
    solver->allocDeviceArrays = allocDeviceArrays_cg2;
    solver->freeDeviceArrays = freeDeviceArrays_cg2;

  } else if (flag == CUDAPCG_MINRES2_SOLVER){
    if (solver->preconditioner_flag == CUDAPCG_TRUE)
      solver->solve = solve_pminres2;
    else
      solver->solve = solve_minres2;
    solver->setX0 = setX0_minres2;
    solver->allocDeviceArrays = allocDeviceArrays_minres2;
    solver->freeDeviceArrays = freeDeviceArrays_minres2;
  }

  if (solver->s != NULL) { HANDLE_ERROR(hipFree(solver->s)); solver->s = NULL; }
  if (solver->userAllocatedArrays_flag && flag == CUDAPCG_MINRES_SOLVER){
    HANDLE_ERROR(hipMalloc(&(solver->s),solver->model->nvars*sizeof(cudapcgVar_t)));
  }

  solver->solver_flag = flag;
  return CUDAPCG_TRUE;
  
}
//------------------------------------------------------------------------------
cudapcgFlag_t setResNorm(cudapcgFlag_t flag){
  if (solver == NULL) return CUDAPCG_FALSE;
  solver->resnorm_flag = flag > CUDAPCG_ERROR_NORM ? CUDAPCG_L2_NORM : flag; // defaults to L2
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
