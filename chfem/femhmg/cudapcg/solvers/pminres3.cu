/*

*/

#include "pminres3.h"
#include "minres3.h"
#include "../kernels/wrappers.h"

//---------------------------------
///////////////////////////////////
//////// PUBLIC FUNCTIONS /////////
///////////////////////////////////
//---------------------------------

cudapcgFlag_t setX0_pminres3(cudapcgSolver_t *solver, cudapcgVar_t *x0, cudapcgFlag_t mustInterpolate){
  return setX0_minres3(solver,x0,mustInterpolate);
}
//------------------------------------------------------------------------------
cudapcgFlag_t allocDeviceArrays_pminres3(cudapcgSolver_t *solver){
  return allocDeviceArrays_minres3(solver);
}
//------------------------------------------------------------------------------
cudapcgFlag_t freeDeviceArrays_pminres3(cudapcgSolver_t *solver){
  return freeDeviceArrays_minres3(solver);
}
//------------------------------------------------------------------------------
cudapcgFlag_t solve_pminres3(cudapcgSolver_t *solver, cudapcgVar_t *res_x){
    if (solver == NULL)
        return CUDAPCG_FALSE;

    solver->count++;

    cudapcgVar_t *x = solver->x;
    cudapcgVar_t *r = solver->r;
    cudapcgVar_t *d = solver->d;

    unsigned int n = solver->model->nvars;
    unsigned int n_stopping_criteria = solver->model->nhmgvars;

    #ifdef CUDAPCG_TRACK_STOPCRIT
    cudapcgVar_t *stopcrit_metrics = (double *)malloc(sizeof(double)*(solver->max_iterations+1));
    #endif
    
    cudapcgModel_t *model = solver->model;

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float time, mean_time=0.0;
    HANDLE_ERROR(hipEventRecord(start,0));

    if (!solver->x0_hasBeenSet_flag)
      zeros(x,n);

    double a, b, delta, delta_0, delta_old, stop_metric, res_0;
    unsigned char mustContinueIterating = 1;

    solver->iteration = 0;

    // residual parameters for x0=[0]
    if (solver->resnorm_flag == CUDAPCG_INF_NORM)
      res_0 = (double) absmax(r,n);
    else
      res_0 = solver->dotPreConditioner(model,r,NULL,1.0);

    solver->applyPreConditioner(model,r,NULL,1.0,0.0,r);  // r = M^-1 * r
    delta_0 = solver->dotAprod(model,r,1.0);              // delta = dot(r,A*r)
    solver->applyinvPreConditioner(model,r,NULL,1.0,0.0,r);  // r = M * r

    // Check if x0=[0] has already satisfied an absolute tolerance
    // This is a safety check. As we perform dimensionless residual evaluation, with
    // respect to delta_0, numerical trouble might occur if this value is too small.
    if (abs_double(res_0) < 0.000000000001){ // 1e-12
        solver->residual = 0.0;
        zeros(x,n); // safety
        HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
        printf("%sNull solution satisfied MINRES.\n",solver->header_str);
        return CUDAPCG_TRUE;
    }

    // check if an initial guess was provided
    if (solver->x0_hasBeenSet_flag){
      // recalculate resiudals considering initial guess
      solver->Aprod(model,x,-1.0,1.0,r);                       // r += -1.0*A*x
      solver->applyPreConditioner(model,r,NULL,1.0,0.0,r);     // r = M^-1 * r
      delta = solver->dotAprod(model,r,1.0);                   // delta = dot(r,A*r)
      solver->applyinvPreConditioner(model,r,NULL,1.0,0.0,r);  // r = M * r
      // Check if initial guess has already satisfied dimensionless tolerance
      if (!isResidualAboveTol(delta,delta_0,solver->num_tol)){
          solver->residual = evalResidual(delta,delta_0);
          HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
          printf("%sInitial guess satisfied MINRES.\n",solver->header_str);
          return CUDAPCG_TRUE;
      }
    } else {
      // assume that x0=[0] is initial guess
      delta = delta_0;
    }
    delta_old = delta;

    #ifdef CUDAPCG_TRACK_STOPCRIT
    switch (solver->resnorm_flag){
      case CUDAPCG_L2_NORM:
        //stop_metric = dotprod(r,r,n);
        stop_metric = solver->dotPreConditioner(model,r,NULL,1.0);
        stop_metric = evalResidual(stop_metric,res_0);
        break;
      case CUDAPCG_INF_NORM:
        stop_metric = ((double) absmax(r,n))/res_0;
        break;
      case CUDAPCG_ERROR_NORM:
        stop_metric = 1.0;
        break;
    }
    stopcrit_metrics[solver->iteration] = stop_metric;
    #endif

    // First iteration outside of while loop
    solver->iteration++;
    solver->applyPreConditioner(model,r,NULL,1.0,0.0,d);         // d = M^-1 * r 
    a = delta / solver->dotPreConditionerA2prod(model,d,1.0);    // a = delta/dot(Ad,M^-1*Ad)
    axpy_iny(x,d, a,n);                                          // x +=  a*d
    solver->Aprod(model,d,-a,1.0,r);                             // r = -a*A*d + r
    solver->applyPreConditioner(model,r,NULL,1.0,0.0,r);         // r = M^-1 * r 
    delta = solver->dotAprod(model,r,1.0);                       // delta = dot(r,A*r)
    solver->applyinvPreConditioner(model,r,NULL,1.0,0.0,r);      // r = M * r (important for stopping criteria)

    switch (solver->resnorm_flag){
      case CUDAPCG_L2_NORM:
        //stop_metric = dotprod(r,r,n);
        stop_metric = solver->dotPreConditioner(model,r,NULL,1.0);
        mustContinueIterating = isResidualAboveTol(stop_metric,res_0,solver->num_tol);
        stop_metric = evalResidual(stop_metric,res_0);
        break;
      case CUDAPCG_INF_NORM:
        stop_metric = ((double) absmax(r,n))/res_0;
        mustContinueIterating = stop_metric > solver->num_tol;
        break;
      case CUDAPCG_ERROR_NORM:
        stop_metric = abs_double(a)*((double)absmax(d,n_stopping_criteria))/((double)absmax(x,n_stopping_criteria));
        mustContinueIterating = stop_metric > solver->num_tol;
        break;
    }
    #ifdef CUDAPCG_TRACK_STOPCRIT
    stopcrit_metrics[solver->iteration] = stop_metric;
    #endif

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
    mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;

    #ifndef CUDAPCG_QUIET_ITERATIONS
      if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
        printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
      else
        printf("\r%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
    #endif

    while (mustContinueIterating && solver->iteration < solver->max_iterations){
        HANDLE_ERROR(hipEventRecord(start,0));

        solver->iteration++;
        b = delta/delta_old;
        solver->applyPreConditioner(model,r,d,1.0,b,d);            // d = M^-1*r+b*d
        a = delta / solver->dotPreConditionerA2prod(model,d,1.0);  // a = delta/dot(Ad,M^-1*Ad)
        axpy_iny(x,d, a,n);                                        // x +=  a*d
        solver->Aprod(model,d,-a,1.0,r);                           // r = -a*A*d + r
        solver->applyPreConditioner(model,r,NULL,1.0,0.0,r);       // r = M^-1 * r 
        delta_old = delta;
        delta = solver->dotAprod(model,r,1.0);                     // delta = dot(r,A*r)
        solver->applyinvPreConditioner(model,r,NULL,1.0,0.0,r);    // r = M * r (important for stopping criteria)

        switch (solver->resnorm_flag){
          case CUDAPCG_L2_NORM:
            //stop_metric = dotprod(r,r,n);
            stop_metric = solver->dotPreConditioner(model,r,NULL,1.0);
            mustContinueIterating = isResidualAboveTol(stop_metric,res_0,solver->num_tol);
            stop_metric = evalResidual(stop_metric,res_0);
            break;
          case CUDAPCG_INF_NORM:
            stop_metric = ((double) absmax(r,n))/res_0;
            mustContinueIterating = stop_metric > solver->num_tol;
            break;
          case CUDAPCG_ERROR_NORM:
            stop_metric = abs_double(a)*((double)absmax(d,n_stopping_criteria))/((double)absmax(x,n_stopping_criteria));
            mustContinueIterating = stop_metric > solver->num_tol;
            break;
        }
        #ifdef CUDAPCG_TRACK_STOPCRIT
        stopcrit_metrics[solver->iteration] = stop_metric;
        #endif

        HANDLE_ERROR(hipEventRecord(stop,0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
        mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;

        #ifndef CUDAPCG_QUIET_ITERATIONS
          if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
            printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
          else
            printf("\r%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
        #endif
    }

    #ifdef CUDAPCG_QUIET_ITERATIONS
      if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
        printf("%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
      else
        printf("%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
    #endif
    printf("\n");

    // Evaluate dimensionless residual
    solver->residual = stop_metric;
    solver->foundSolution_flag = solver->residual <= solver->num_tol;

    // Copy result back to cpu
    HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));

    solver->mean_time_per_iteration = mean_time*0.001; // value is in ms

    #ifdef CUDAPCG_TRACK_STOPCRIT
    char filename_buffer[2048];
    sprintf(filename_buffer,"%s_stopcrit_metrics_%lu.bin",model->name,solver->count);
    FILE * file = fopen(filename_buffer,"wb");
    if (file)
      fwrite(stopcrit_metrics,sizeof(double)*(solver->iteration+1),1,file);
    fclose(file);
    free(stopcrit_metrics);
    #endif

    return solver->foundSolution_flag;
}
//------------------------------------------------------------------------------
