/*

*/

#include "minres2.h"
#include "xreduce.h"
#include "../kernels/wrappers.h"

//---------------------------------
///////////////////////////////////
//////// PUBLIC FUNCTIONS /////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setX0_minres2(cudapcgSolver_t *solver, cudapcgVar_t *x0, cudapcgFlag_t mustInterpolate){
  size_t var_sz = sizeof(cudapcgVar_t)*((size_t)solver->model->nvars);
  if (solver->d == NULL) HANDLE_ERROR(hipMalloc(&solver->d,var_sz));
  if (mustInterpolate){
      unsigned int nodal_dofs = solver->model->nvars/solver->model->nelem;
      size_t coarse_var_sz = sizeof(cudapcgVar_t)*nodal_dofs*((solver->model->ncols)/2)*((solver->model->nrows)/2)*((solver->model->nlayers)/2+(solver->model->nlayers<2));
      cudapcgVar_t *temp=NULL;
      HANDLE_ERROR(hipMalloc(&temp,coarse_var_sz));
      HANDLE_ERROR(hipMemcpy(temp,x0,coarse_var_sz,hipMemcpyHostToDevice));
      interpl2(temp,solver->model->nrows,solver->model->ncols,solver->model->nlayers,solver->model->nvars/solver->model->nelem,solver->d);
      HANDLE_ERROR(hipFree(temp));
  } else {
      HANDLE_ERROR(hipMemcpy(solver->d,x0,var_sz,hipMemcpyHostToDevice));
  }
  solver->x0_hasBeenSet_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t allocDeviceArrays_minres2(cudapcgSolver_t *solver){
  size_t sz = sizeof(cudapcgVar_t)*solver->model->nvars;
  if (!solver->x0_hasBeenSet_flag) HANDLE_ERROR(hipMalloc(&solver->d,sz));
  // allocate arrays that will be used to store dotprod kernel results (within cudapcg_kernels.h)
  allocDotProdArrs(solver->model->nvars);
  solver->userAllocatedArrays_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t freeDeviceArrays_minres2(cudapcgSolver_t *solver){
  if (!(solver->x0_hasBeenSet_flag)){
    if (solver->d!=NULL) HANDLE_ERROR(hipFree(solver->d));
    solver->d = NULL;
  }
  freeDotProdArrs();
  solver->userAllocatedArrays_flag = CUDAPCG_FALSE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t solve_minres2(cudapcgSolver_t *solver, cudapcgVar_t *res_x){
    if (solver == NULL)
        return CUDAPCG_FALSE;

    solver->count++;

    cudapcgVar_t *r = solver->r;
    cudapcgVar_t *d = solver->d;

    unsigned int n = solver->model->nvars;
    unsigned int n_stopping_criteria = solver->model->nhmgvars;

    #ifdef CUDAPCG_TRACK_STOPCRIT
    cudapcgVar_t *stopcrit_metrics = (double *)malloc(sizeof(double)*(solver->max_iterations+1));
    #endif
    
    cudapcgModel_t *model = solver->model;

    double **res_per_it = (double **)malloc(sizeof(double*)*model->nvarspernode);
    res_per_it[0] = (double *)malloc(sizeof(double)*(solver->max_iterations+1)*model->nvarspernode);
    for (int ii=1; ii<model->nvarspernode; ii++) res_per_it[ii] = &(res_per_it[0][ii*(solver->max_iterations+1)]);

    // double **norms = NULL;
    // if (solver->xreduce_flag == CUDAPCG_XREDUCE_NONE){
    //   norms = (double **)malloc(sizeof(double *)*4);
    //   norms[0] = (double *)malloc(sizeof(double)*(solver->max_iterations+1)*4);
    //   norms[1] = &(norms[0][1*(solver->max_iterations+1)]);
    //   norms[2] = &(norms[0][2*(solver->max_iterations+1)]);
    //   norms[3] = &(norms[0][3*(solver->max_iterations+1)]);
    //   norms[0][0]=0.0;
    //   norms[1][0]=0.0;
    //   norms[2][0]=0.0;
    //   norms[3][0]=0.0;
    // }

    cudapcgVar_t *x=NULL;
    if (solver->xreduce_flag == CUDAPCG_XREDUCE_ONLYDIR){
      HANDLE_ERROR(hipMalloc(&x,sizeof(cudapcgVar_t)*model->nhmgvars/model->nvarspernode));
      zeros(x,model->nhmgvars/model->nvarspernode);
    } else if (solver->xreduce_flag == CUDAPCG_XREDUCE_FULL){
      HANDLE_ERROR(hipMalloc(&x,sizeof(cudapcgVar_t)*model->nhmgvars));
      zeros(x,model->nhmgvars);
    }

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float time, mean_time=0.0;
    HANDLE_ERROR(hipEventRecord(start,0));

    double a, b, delta, delta_0, delta_old, stop_metric, res_0;
    unsigned char mustContinueIterating = 1;

    solver->iteration = 0;

    // Init resulting effective fields
    for (int ii=0; ii<model->nvarspernode; ii++) res_per_it[ii][0]=0.0;

    // residual parameters for x0=[0]
    delta_0 = solver->dotAprod(model,r,1.0);  // delta = dot(r,A*r)

    if (solver->resnorm_flag == CUDAPCG_INF_NORM)
      res_0 = (double) absmax(r,n);//(scl_factor*scl_factor)*((double) absmax(r,n));
    else
      res_0 = dotprod(r,r,n);//(scl_factor*scl_factor)*dotprod(r,r,n); //delta_0;

    // Check if x0=[0] has already satisfied an absolute tolerance
    // This is a safety check. As we perform dimensionless residual evaluation, with
    // respect to delta_0, numerical trouble might occur if this value is too small.
    if (abs_double(res_0) < 0.000000000001){ // 1e-12
        solver->residual = 0.0;
        printf("%sNull solution satisfied MINRES.\n",solver->header_str);
        return CUDAPCG_TRUE;
    }

    // check if an initial guess was provided
    if (solver->x0_hasBeenSet_flag){
      // recalculate resiudals considering initial guess
      solver->Aprod(model,d,-1.0,1.0,r);             // r += -1.0*A*x (d is being used to store x0)
      delta = solver->dotAprod(model,r,1.0);         // delta = dot(r,A*r)
      switch (solver->xreduce_flag){
        case CUDAPCG_XREDUCE_ONLYDIR:
          axpy_iny_with_stride(x,d,1.0,model->nhmgvars,model->nvarspernode,solver->xreduce_shift);
          break;
        case CUDAPCG_XREDUCE_FULL:
          arrcpy(d,model->nhmgvars,x);
          break;
      }
      // update effective fields
      for (int ii=0; ii<model->nvarspernode; ii++) res_per_it[ii][0]=reduce_with_stride(d,model->nhmgvars,model->nvarspernode,ii);
      // Check if initial guess has already satisfied dimensionless tolerance
      if (!isResidualAboveTol(delta,delta_0,solver->num_tol)){
          solver->residual = evalResidual(delta,delta_0);
          printf("%sInitial guess satisfied MINRES.\n",solver->header_str);
          return CUDAPCG_TRUE;
      }
    } else {
      // assume that x0=[0] is initial guess
      delta = delta_0;
    }
    delta_old = delta;

    arrcpy(r,n,d);                       // d = r

    #ifdef CUDAPCG_TRACK_STOPCRIT
    switch (solver->resnorm_flag){
      case CUDAPCG_L2_NORM:
        stop_metric = dotprod(r,r,n);
        stop_metric = evalResidual(stop_metric,res_0);
        break;
      case CUDAPCG_INF_NORM:
        stop_metric = ((double) absmax(r,n))/res_0;
        break;
      case CUDAPCG_ERROR_NORM:
        stop_metric = 1.0;
        break;
    }
    stopcrit_metrics[solver->iteration] = stop_metric;
    #endif

    // First iteration outside of while loop
    solver->iteration++;
    a = delta / solver->dotA2prod(model,d,1.0);    // a = delta/(Ad*Ad)
    update_xreduce(model,solver->iteration,solver->xreduce_flag,solver->xreduce_shift,solver->reduce_stab_factor,a,d,x,res_per_it);
    solver->Aprod(model,d,-a,1.0,r);               // r = -a*A*d + r
    delta = solver->dotAprod(model,r,1.0);         // delta = dot(r,A*r)

    switch (solver->resnorm_flag){
      case CUDAPCG_L2_NORM:
        stop_metric = dotprod(r,r,n);
        mustContinueIterating = isResidualAboveTol(stop_metric,res_0,solver->num_tol);
        stop_metric = evalResidual(stop_metric,res_0);
        break;
      case CUDAPCG_INF_NORM:
        stop_metric = ((double) absmax(r,n))/res_0;
        mustContinueIterating = stop_metric > solver->num_tol;
        break;
      case CUDAPCG_ERROR_NORM:
        stop_metric = abs_double(a)*((double)absmax(d,n_stopping_criteria))/1.0;//((double)absmax(x,n_stopping_criteria));
        mustContinueIterating = stop_metric > solver->num_tol;
        break;
    }
    #ifdef CUDAPCG_TRACK_STOPCRIT
    stopcrit_metrics[solver->iteration] = stop_metric;
    #endif

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
    mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;

    #ifndef CUDAPCG_QUIET_ITERATIONS
      if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
        printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
      else
        printf("\r%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
    #endif

    while (mustContinueIterating && solver->iteration < solver->max_iterations){
        HANDLE_ERROR(hipEventRecord(start,0));

        solver->iteration++;
        b = delta/delta_old;
        axpy(r,d,b,n,d);                              // d = r+b*d
        a = delta / solver->dotA2prod(model,d,1.0);   // a = delta/(Ad*Ad)
        update_xreduce(model,solver->iteration,solver->xreduce_flag,solver->xreduce_shift,solver->reduce_stab_factor,a,d,x,res_per_it);
        solver->Aprod(model,d,-a,1.0,r);              // r = -a*A*d + r
        delta_old = delta;
        delta = solver->dotAprod(model,r,1.0);        // delta = dot(r,A*r)

        switch (solver->resnorm_flag){
          case CUDAPCG_L2_NORM:
            stop_metric = dotprod(r,r,n);
            mustContinueIterating = isResidualAboveTol(stop_metric,res_0,solver->num_tol);
            stop_metric = evalResidual(stop_metric,res_0);
            break;
          case CUDAPCG_INF_NORM:
            stop_metric = ((double) absmax(r,n))/res_0;
            mustContinueIterating = stop_metric > solver->num_tol;
            break;
          case CUDAPCG_ERROR_NORM:
            stop_metric = abs_double(a)*((double)absmax(d,n_stopping_criteria))/1.0;//((double)absmax(x,n_stopping_criteria));
            mustContinueIterating = stop_metric > solver->num_tol;
            break;
        }
        #ifdef CUDAPCG_TRACK_STOPCRIT
        stopcrit_metrics[solver->iteration] = stop_metric;
        #endif

        HANDLE_ERROR(hipEventRecord(stop,0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
        mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;

        #ifndef CUDAPCG_QUIET_ITERATIONS
          if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
            printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
          else
            printf("\r%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
        #endif
    }

    #ifdef CUDAPCG_QUIET_ITERATIONS
      if (solver->resnorm_flag != CUDAPCG_ERROR_NORM)
        printf("%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,stop_metric);
      else
        printf("%siteration: %d, residual: %.3e, error: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0),stop_metric);
    #endif
    printf("\n");

    // Evaluate dimensionless residual
    solver->residual = stop_metric;
    solver->foundSolution_flag = solver->residual <= solver->num_tol;

    solver->mean_time_per_iteration = mean_time*0.001; // value is in ms

    char filename_buffer[2048];
    sprintf(filename_buffer,"%s_xreduce_metrics_%lu.bin",model->name,solver->count);
    save_xreduce(&filename_buffer[0],model,solver->iteration,solver->xreduce_flag,solver->xreduce_shift,solver->xreduce_scale,res_per_it);
    
    double *ptr=NULL;
    for (int ii=0; ii<model->nvarspernode; ii++){
      ptr = res_per_it[ii];
      switch (solver->xreduce_flag){
        case CUDAPCG_XREDUCE_NONE:
          for (int jj=solver->iteration; jj>0; jj--) ptr[jj-1]+=ptr[jj];
          res_x[ii] = *ptr;
          break;
        case CUDAPCG_XREDUCE_ONLYDIR:
          res_x[ii] = solver->xreduce_shift == ii ? ptr[solver->iteration] : 0.0;
          break;
        case CUDAPCG_XREDUCE_FULL:
          res_x[ii] = ptr[solver->iteration];
          break;
      }
    }
    free(res_per_it[0]);
    free(res_per_it);

    // FILE * file = NULL;
    // if (norms){
    //   sprintf(filename_buffer,"%s_absmax_metrics_%lu.bin",model->name,solver->count);
    //   file = fopen(filename_buffer,"wb");
    //   if (file) fwrite(norms[0],sizeof(double)*(solver->iteration+1),1,file);
    //   fclose(file);

    //   sprintf(filename_buffer,"%s_l2norm_metrics_%lu.bin",model->name,solver->count);
    //   file = fopen(filename_buffer,"wb");
    //   if (file) fwrite(norms[1],sizeof(double)*(solver->iteration+1),1,file);
    //   fclose(file);

    //   sprintf(filename_buffer,"%s_alpha_metrics_%lu.bin",model->name,solver->count);
    //   file = fopen(filename_buffer,"wb");
    //   if (file) fwrite(norms[2],sizeof(double)*(solver->iteration+1),1,file);
    //   fclose(file);

    //   sprintf(filename_buffer,"%s_dreduce_metrics_%lu.bin",model->name,solver->count);
    //   file = fopen(filename_buffer,"wb");
    //   if (file) fwrite(norms[3],sizeof(double)*(solver->iteration+1),1,file);
    //   fclose(file);

    //   free(norms[0]);
    //   free(norms);
    //   norms=NULL;
    // }

    if (solver->xreduce_flag > CUDAPCG_XREDUCE_NONE) HANDLE_ERROR(hipFree(x)); x = NULL;

    #ifdef CUDAPCG_TRACK_STOPCRIT
    sprintf(filename_buffer,"%s_stopcrit_metrics_%lu.bin",model->name,solver->count);
    FILE * file = fopen(filename_buffer,"wb");
    if (file)
      fwrite(stopcrit_metrics,sizeof(double)*(solver->iteration+1),1,file);
    fclose(file);
    free(stopcrit_metrics);
    #endif

    return solver->foundSolution_flag;
}
//------------------------------------------------------------------------------
