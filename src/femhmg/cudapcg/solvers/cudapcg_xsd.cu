/*

*/

#include "cudapcg_xsd.h"
#include "../kernels/cudapcg_kernels_wrappers.h"

//---------------------------------
///////////////////////////////////
//////// PUBLIC FUNCTIONS /////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setX0_XSD(cudapcgSolver_t *solver, cudapcgVar_t *x0, cudapcgFlag_t mustInterpolate){
  unsigned int var_sz = sizeof(cudapcgVar_t)*solver->model->nvars;
  if (solver->x == NULL)
      HANDLE_ERROR(hipMalloc(&solver->x,var_sz));
  if (mustInterpolate){
      if (solver->d == NULL)
          HANDLE_ERROR(hipMalloc(&solver->d,var_sz));
      unsigned int nodal_dofs = solver->model->nvars/solver->model->nelem;
      unsigned int coarse_var_sz = sizeof(cudapcgVar_t)*nodal_dofs*((solver->model->ncols)/2)*((solver->model->nrows)/2)*((solver->model->nlayers)/2+(solver->model->nlayers<2));
      HANDLE_ERROR(hipMemcpy(solver->d,x0,coarse_var_sz,hipMemcpyHostToDevice));
      interpl2(solver->x,solver->d,solver->model->ncols,solver->model->nrows,solver->model->nlayers,solver->model->nvars/solver->model->nelem);
  } else {
      HANDLE_ERROR(hipMemcpy(solver->x,x0,var_sz,hipMemcpyHostToDevice));
  }
  solver->x0_hasBeenSet_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t allocDeviceArrays_XSD(cudapcgSolver_t *solver){
  unsigned int sz = sizeof(cudapcgVar_t)*solver->model->nvars;
  if (!solver->x0_hasBeenSet_flag)
      HANDLE_ERROR(hipMalloc(&solver->x,sz));
  HANDLE_ERROR(hipMalloc(&solver->d,sz));
  // allocate arrays that will be used to store dotprod kernel results (within cudapcg_kernels.h)
  allocDotProdArrs(solver->model->nvars);
  solver->userAllocatedArrays_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t freeDeviceArrays_XSD(cudapcgSolver_t *solver){
  if (!(solver->x0_hasBeenSet_flag)){
      if (solver->x!=NULL) HANDLE_ERROR(hipFree(solver->x));
      solver->x = NULL;
  }
  if (solver->d!=NULL) HANDLE_ERROR(hipFree(solver->d)); solver->d = NULL;
  freeDotProdArrs();
  solver->userAllocatedArrays_flag = CUDAPCG_FALSE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t solve_XSD(cudapcgSolver_t *solver, cudapcgVar_t *res_x){
    if (solver == NULL)
        return CUDAPCG_FALSE;
        
    cudapcgVar_t *x = solver->x;
    cudapcgVar_t *s = solver->r;
    cudapcgVar_t *d = solver->d;
    
    unsigned int n = solver->model->nvars;
    
    cudapcgModel_t *model = solver->model;
        
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float time, mean_time=0.0;
    HANDLE_ERROR(hipEventRecord(start,0));

    if (!solver->x0_hasBeenSet_flag)
      zeros(x,n);

    cudapcgVar_t a, delta, delta_0, delta_old;
    
    solver->iteration = 0;

    // residual parameters for x0=[0]
    solver->applyPreConditioner(model,s,s,0.0,s);  // s = M^-1*r + 0.0*r (s=r on preprocessing)
    delta_0 = solver->dotprod_precond(model,s);    // delta = (M*s)*s
    
    // Check if x0=[0] has already satisfied an absolute tolerance
    // This is a safety check. As we perform dimensionless residual evaluation,
    // in respect to delta_0, numerical trouble might occur if this value is too small.
    if (ABS(delta_0) < 0.000000000001){ // 1e-12
        solver->residual = 0.0;
        // Copy result back to cpu
        HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
        printf("%sNull solution satisfied PCG.\n",solver->header_str);
        return CUDAPCG_TRUE;
    }

    // check if an initial guess was provided
    if (solver->x0_hasBeenSet_flag){
      // recalculate resiudals considering initial guess
      solver->Aprod(model,x,s,-1.0,CUDAPCG_TRUE);   // s += -(M^-1*A*x)
      delta = solver->dotprod_precond(model,s);     // delta = (M*s)*s
      // Check if initial guess has already satisfied dimensionless tolerance
      if (!isResidualAboveTol(delta,delta_0,solver->num_tol)){
          solver->residual = evalResidual(delta,delta_0);
          // Copy result back to cpu
          HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
          printf("%sInitial guess satisfied PCG.\n",solver->header_str);
          return CUDAPCG_TRUE;
      }
    } else {
      // assume that x0=[0] is initial guess
      delta = delta_0;
    }
    delta_old = delta;
    arrcpy(d,s,n);     // d = s

    // First iteration outside of while loop
    // ATTENTION: d=s on first iteration
    solver->iteration++;
    a = delta / solver->dotprod_Aprod(model,d); // a = delta/(d*(A*d))
    sumVecIntoFirst(x,d,a,n);                   // x += a*d
    solver->Aprod(model,d,s,-a,CUDAPCG_TRUE);   // s += -a*(M^-1*A*d)
    delta = solver->dotprod_precond(model,s);   // delta = (M*s)*s
    
    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
    mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;
    
    #ifndef CUDAPCG_QUIET_ITERATIONS
      printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
    #endif

    while (isResidualAboveTol(delta,delta_0,solver->num_tol) && solver->iteration < solver->max_iterations){
        HANDLE_ERROR(hipEventRecord(start,0));
        
        solver->iteration++;
        sumVec(s,d,delta/delta_old,n,d);             // d = s+(delta/delta_old)*d
        a = delta / solver->dotprod_Aprod(model,d);  // a = delta/(d*(A*d))
        sumVecIntoFirst(x,d,a,n);                    // x += a*d
        solver->Aprod(model,d,s,-a,CUDAPCG_TRUE);    // s += -a*(M^-1*A*d)
        delta_old = delta;
        delta = solver->dotprod_precond(model,s);    // delta = (M*s)*s
        
        HANDLE_ERROR(hipEventRecord(stop,0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
        mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;
        
        #ifndef CUDAPCG_QUIET_ITERATIONS
          printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
        #endif
    }
    
    #ifdef CUDAPCG_QUIET_ITERATIONS
      printf("%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
    #endif
    printf("\n");
    
    // Evaluate dimensionless residual
    solver->residual = evalResidual(delta,delta_0);
    
    // Copy result back to cpu
    HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
    
    solver->mean_time_per_iteration = mean_time*0.001; // value is in ms
    
    solver->foundSolution_flag = !isResidualAboveTol(delta,delta_0,solver->num_tol);

    return solver->foundSolution_flag;
}
//------------------------------------------------------------------------------

