/*

*/

#include "cudapcg_default.h"
#include "../kernels/cudapcg_kernels_wrappers.h"

//---------------------------------
///////////////////////////////////
//////// PUBLIC FUNCTIONS /////////
///////////////////////////////////
//---------------------------------

//------------------------------------------------------------------------------
cudapcgFlag_t setX0_default(cudapcgSolver_t *solver, cudapcgVar_t *x0, cudapcgFlag_t mustInterpolate){
  unsigned int var_sz = sizeof(cudapcgVar_t)*solver->model->nvars;
  if (solver->x == NULL)
      HANDLE_ERROR(hipMalloc(&solver->x,var_sz));
  if (mustInterpolate){
      if (solver->q == NULL)
          HANDLE_ERROR(hipMalloc(&solver->q,var_sz));
      unsigned int nodal_dofs = solver->model->nvars/solver->model->nelem;
      unsigned int coarse_var_sz = sizeof(cudapcgVar_t)*nodal_dofs*((solver->model->ncols)/2)*((solver->model->nrows)/2)*((solver->model->nlayers)/2+(solver->model->nlayers<2));
      HANDLE_ERROR(hipMemcpy(solver->q,x0,coarse_var_sz,hipMemcpyHostToDevice));
      interpl2(solver->x,solver->q,solver->model->ncols,solver->model->nrows,solver->model->nlayers,solver->model->nvars/solver->model->nelem);
  } else {
      HANDLE_ERROR(hipMemcpy(solver->x,x0,var_sz,hipMemcpyHostToDevice));
  }
  solver->x0_hasBeenSet_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t allocDeviceArrays_default(cudapcgSolver_t *solver){
  unsigned int sz = sizeof(cudapcgVar_t)*solver->model->nvars;
  if (!solver->x0_hasBeenSet_flag)
      HANDLE_ERROR(hipMalloc(&solver->x,sz));
  HANDLE_ERROR(hipMalloc(&solver->d,sz));
  if (solver->q == NULL)
      HANDLE_ERROR(hipMalloc(&solver->q,sz));
  if (solver->mustAssemblePreConditioner){
      allocPreConditioner(solver->model);
      solver->assemblePreConditioner(solver->model);
  }
  // allocate arrays that will be used to store dotprod kernel results (within cudapcg_kernels.h)
  allocDotProdArrs(solver->model->nvars);
  solver->userAllocatedArrays_flag = CUDAPCG_TRUE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t freeDeviceArrays_default(cudapcgSolver_t *solver){
  if (!(solver->x0_hasBeenSet_flag)){
      if (solver->x!=NULL) HANDLE_ERROR(hipFree(solver->x));
      solver->x = NULL;
  }
  if (solver->d!=NULL) HANDLE_ERROR(hipFree(solver->d)); solver->d = NULL;
  if (solver->q!=NULL) HANDLE_ERROR(hipFree(solver->q)); solver->q = NULL;
  if (solver->mustAssemblePreConditioner)
      freePreConditioner();
  freeDotProdArrs();
  solver->userAllocatedArrays_flag = CUDAPCG_FALSE;
  return CUDAPCG_TRUE;
}
//------------------------------------------------------------------------------
cudapcgFlag_t solve_default(cudapcgSolver_t *solver, cudapcgVar_t *res_x){
    if (solver == NULL)
        return CUDAPCG_FALSE;
        
    cudapcgVar_t *x = solver->x;
    cudapcgVar_t *r = solver->r;
    cudapcgVar_t *d = solver->d;
    cudapcgVar_t *q = solver->q;
    
    unsigned int n = solver->model->nvars;
    
    cudapcgModel_t *model = solver->model;
        
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float time, mean_time=0.0;
    HANDLE_ERROR(hipEventRecord(start,0));

    if (!solver->x0_hasBeenSet_flag)
      zeros(x,n);
    zeros(q,n);

    cudapcgVar_t a, delta, delta_0, delta_old;
    
    solver->iteration = 0;
    
    // residual parameters for x0=[0]    
    solver->applyPreConditioner(model,r,r,0.0,d);  // s = M^-1 * r (d=s on first iteration)
    delta_0 = dotprod(r,d,n);                      // delta = r*s
    
    // Check if x0=[0] has already satisfied an absolute tolerance
    // This is a safety check. As we perform dimensionless residual evaluation, with
    // respect to delta_0, numerical trouble might occur if this value is too small.
    if (ABS(delta_0) < 0.000000000001){ // 1e-12
        solver->residual = 0.0;
        // Copy result back to cpu
        HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
        printf("%sNull solution satisfied PCG.\n",solver->header_str);
        return CUDAPCG_TRUE;
    }
    
    // check if an initial guess was provided
    if (solver->x0_hasBeenSet_flag){
      // recalculate resiudals considering initial guess
      solver->Aprod(model,x,q,1.0,CUDAPCG_FALSE);    // q = A*x
      sumVecIntoFirst(r,q,-1.0,n);                   // r += -q
      solver->applyPreConditioner(model,r,r,0.0,d);  // s = M^-1 * r (d=s on first iteration)
      delta = dotprod(r,d,n);                        // delta = r*s
      // Check if initial guess has already satisfied dimensionless tolerance
      if (!isResidualAboveTol(delta,delta_0,solver->num_tol)){
          solver->residual = evalResidual(delta,delta_0);
          // Copy result back to cpu
          HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));
          printf("%sInitial guess satisfied PCG.\n",solver->header_str);
          return CUDAPCG_TRUE;
      }
    } else {
      // assume that x0=[0] is initial guess
      delta = delta_0;
    }
    delta_old = delta;

    // First iteration outside of while loop
    solver->iteration++;
    solver->Aprod(model,d,q,1.0,CUDAPCG_FALSE);    // q = A*d    
    a = delta / dotprod(d,q,n);                    // a = delta/(d*q)
    sumVecIntoFirst(x,d,a,n);                      // x += a*d
    sumVecIntoFirst(r,q,-a,n);                     // r += -a*q
    solver->applyPreConditioner(model,r,r,0.0,q);  // s = M^-1 * r (use q to store s)
    delta = dotprod(r,q,n);                        // delta = r*s
    
    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
    mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;
    
    #ifndef CUDAPCG_QUIET_ITERATIONS
      printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
    #endif

    while (isResidualAboveTol(delta,delta_0,solver->num_tol) && solver->iteration < solver->max_iterations){
        HANDLE_ERROR(hipEventRecord(start,0));
        
        solver->iteration++;
        sumVec(q,d,delta/delta_old,n,d);              // d = s+(delta/delta_old)*d
        solver->Aprod(model,d,q,1.0,CUDAPCG_FALSE);   // q = A*d
        a = delta / dotprod(d,q,n);                   // a = delta/(d*q)
        sumVecIntoFirst(x,d,a,n);                     // x += a*p
        sumVecIntoFirst(r,q,-a,n);                    // r += -a*q
        solver->applyPreConditioner(model,r,r,0.0,q); // s = M^-1 * r (use q to store s)
        delta_old = delta;
        delta = dotprod(r,q,n);                       // delta = r*s
        
        HANDLE_ERROR(hipEventRecord(stop,0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        HANDLE_ERROR(hipEventElapsedTime(&time,start,stop));
        mean_time = ((solver->iteration-1)*mean_time + time) / solver->iteration;
        
        #ifndef CUDAPCG_QUIET_ITERATIONS
          printf("\r%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
        #endif
    }
    
    #ifdef CUDAPCG_QUIET_ITERATIONS
      printf("%siteration: %d, residual: %.3e",solver->header_str,solver->iteration,evalResidual(delta,delta_0));
    #endif
    printf("\n");

    // Evaluate dimensionless residual
    solver->residual = evalResidual(delta,delta_0);
    
    // Copy result back to cpu
    HANDLE_ERROR(hipMemcpy(res_x,x,n*sizeof(cudapcgVar_t),hipMemcpyDeviceToHost));

    solver->mean_time_per_iteration = mean_time*0.001; // value is in ms
    
    solver->foundSolution_flag = !isResidualAboveTol(delta,delta_0,solver->num_tol);

    return solver->foundSolution_flag;
}
//------------------------------------------------------------------------------

